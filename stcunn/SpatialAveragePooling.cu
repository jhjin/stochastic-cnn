#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

template <typename Dtype, bool COUNT_INCLUDE_PAD>
__global__ void AvePoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    const int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    const Dtype* const bottom_slice = bottom_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_slice[h * width + w];
      }
    }
    if(COUNT_INCLUDE_PAD)
      top_data[index] = aveval / pool_size;
    else
      top_data[index] = aveval / ((hend - hstart) * (wend - wstart));
  }
}


void THNN_CudaSpatialAveragePooling_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output, int kW, int kH, int dW, int dH, int padW, int padH, bool ceil_mode, bool count_include_pad)
{
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols, nOutputRows;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  THArgCheck(nInputCols >= kW - 2*padW && nInputRows >= kH - 2*padH, 2, "input image smaller than kernel size");
  THArgCheck(kW/2 >= padW && kH/2 >= padH, 2, "pad should be smaller than half of kernel size");

  if(ceil_mode) {
    nOutputCols = ceil(float(nInputCols - kW + 2*padW) / float(dW)) + 1;
    nOutputRows = ceil(float(nInputRows - kH + 2*padH) / float(dH)) + 1;
  }
  else {
    nOutputCols = floor(float(nInputCols - kW + 2*padW) / float(dW)) + 1;
    nOutputRows = floor(float(nInputRows - kH + 2*padH) / float(dH)) + 1;
  }
  if (padW || padH)
  {
    // ensure that the last pooling starts inside the image
    // needed to avoid problems in ceil mode
    if ((nOutputRows - 1)*dH >= nInputRows + padH)
      --nOutputRows;
    if ((nOutputCols  - 1)*dW >= nInputCols  + padW)
      --nOutputCols;
  }

  input = THCudaTensor_newContiguous(state, input);
  float* input_data = THCudaTensor_data(state, input);

  THCudaTensor_resize4d(state, output, batchSize, nInputPlane, nOutputRows, nOutputCols);

  float* output_data = THCudaTensor_data(state, output);

  int count = THCudaTensor_nElement(state, output);

  if(count_include_pad)
    AvePoolForward<float, true>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
        count, input_data,
        batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
        kH, kW, dH, dW, padH, padW, output_data);
  else
    AvePoolForward<float, false>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
        count, input_data,
        batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
        kH, kW, dH, dW, padH, padW, output_data);

  if(input->nDimension == 3)
    THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);

  THCudaTensor_free(state, input);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialAveragePooling.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 2;
}
